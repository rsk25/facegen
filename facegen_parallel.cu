#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>

#include "facegen.h"


#define CHECK_CUDA(err) \
  do { \
    hipError_t CHECK_CUDA_err = (err); \
    if (CHECK_CUDA_err != hipSuccess) { \
      printf("[%s:%d] CUDA error %d (%s)\n", __FILE__, __LINE__, CHECK_CUDA_err, hipGetErrorString(CHECK_CUDA_err)); \
      exit(EXIT_FAILURE); \
    } \
  } while (0)


/*
 * TODO
 * Define global variables here.
 */
extern num_to_gen;

//gpu_mem ptrs for network, inputs, and outputs
static int NETWORK_SIZE_IN_BYTES = 20549132;
static float* gpu_network;
static float* gpu_inputs;  
static float* gpu_outputs;


void facegen_init() {
  /*
   * TODO
   * Initialize required CUDA objects. For example,
   * hipMalloc(...)
   */
	CHECK_CUDA(hipMalloc(&gpu_network, NETWORK_SIZE_IN_BYTES * sizeof(float));
	CHECK_CUDA(hipMalloc(&gpu_input,num_to_gen * 100 * sizeof(float)));
	CHECK_CUDA(hipMalloc(&gpu_output,num_to_gen * 64*64*3 * sizeof(float)));
}

// data-parallelism w.r.t K (col. dim of output of proj)
__global__ void proj(float *in, float *out, float *weight, float *bias, int C, K){
	int k = blockDim.x * blockIdx.x + threadIdx.x;
	if (k >= K) return;
	
	float s = 0;
	for (int c = 0; c<C; c++){
		s += in[c] * weight[c]
	}
	s += bias[k];
	out[k] = s;
}

__global__ void batch_norm(float *inout, float *beta, float *gamma, float *mean, float *var, int HW, int C){		
	int hw = blockDim.x * blockIdx.x + threadIdx.x;
	if (hw >= HW) return;
	
	for (int c = 0; c < C; c++){
		float scaled_gamma = gamma[c] / sqrtf(var[c] + 1e-5);
		inout[hw * C + c] = scaled_gammma * inout[hw * C + c] + (beta[c] - scaled_gammma * mean[c]);
	}
}

__global__ void tanh_layer(float *inout, int HWC){
	int hwc = blockDim.x * blockIdx.x + threadIdx.x;
	if (hwc >= HWC) return;

	inout[hwc] = tanhf(inout[hwc]);
}

__global__ void relu(float *inout, int HWC){
	int hwc = blockDim.x * blockIdx.x + threadIdx.x;
	if (hwc >= HWC) return;

	inout[hwc] = fmaxf(inout[hwc], 0);
}

__global__ void tconv(float *in, float *out, float *weight, float* bias, int H_IN, int W_IN, int C, int K){
	
}


void facegen(int num_to_gen, float *network, float *inputs, float *outputs) {
  /*
   * TODO
   * Implement facegen computation here.
   * See "facegen_seq.c" if you don't know what to do.
   *
   * Below functions should be implemented in here:
   * Host-to-devie memory copy,
   * CUDA kernel launch,
   * Device-to-host memory copy
   */
	float *proj_w = gpu_network; gpu_network += 100 * 8192;
  float *proj_b = gpu_network; gpu_network += 8192;
  float *bn0_beta = gpu_network; gpu_network += 512;
  float *bn0_gamma = gpu_network; gpu_network += 512;
  float *bn0_mean = gpu_network; gpu_network += 512;
  float *bn0_var = gpu_network; gpu_network += 512;
  float *tconv1_w = gpu_network; gpu_network += 5 * 5 * 256 * 512;
  float *tconv1_b = gpu_network; gpu_network += 256;
  float *bn1_beta = gpu_network; gpu_network += 256;
  float *bn1_gamma = gpu_network; gpu_network += 256;
  float *bn1_mean = gpu_network; gpu_network += 256;
  float *bn1_var = gpu_network; gpu_network += 256;
  float *tconv2_w = gpu_network; gpu_network += 5 * 5 * 128 * 256;
  float *tconv2_b = gpu_network; gpu_network += 128;
  float *bn2_beta = gpu_network; gpu_network += 128;
  float *bn2_gamma = gpu_network; gpu_network += 128;
  float *bn2_mean = gpu_network; gpu_network += 128;
  float *bn2_var = gpu_network; gpu_network += 128;
  float *tconv3_w = gpu_network; gpu_network += 5 * 5 * 64 * 128;
  float *tconv3_b = gpu_network; gpu_network += 64;
  float *bn3_beta = gpu_network; gpu_network += 64;
  float *bn3_gamma = gpu_network; gpu_network += 64;
  float *bn3_mean = gpu_network; gpu_network += 64;
  float *bn3_var = gpu_network; gpu_network += 64;
  float *tconv4_w = gpu_network; gpu_network += 5 * 5 * 3 * 64;
  float *tconv4_b = gpu_network; gpu_network += 3;
	
	for (int n = 0; n < num_to_gen; n++){
		
		/* Add MPI_Send, MPI_Recv here*/ 

		// Linear projection layer
		dim3 gridDim();
		dim3 blockDim();
		proj<<<>>>(gpu_input, gpu_fm0, proj_w, proj_b, 100, 8192);
		batch_norm<<<>>>(gpu_input, gpu_fm0, proj_w, proj_b, 100, 8192);
		relu<<<>>>(gpu_input, gpu_fm0, proj_w, proj_b, 100, 8192);

		
	}


}

void facegen_fin() {
  /*
   * TODO
   * Finalize required CUDA objects. For example,
   * hipFree(...)
   */
	CHECK_CUDA(gpu_network);
	CHECK_CUDA(gpu_input);
	CHECK_CUDA(gpu_output);

}
